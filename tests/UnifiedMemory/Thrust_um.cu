#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream> 
#include <hip/hip_runtime.h>
#include <algorithm>
#include "type.h"

using namespace std;

int main(void)
{
    uint64_t number_of_elements = 1024L*1024*1024;
    ulong2 *h_key_array;

    hipMallocManaged(&h_key_array, number_of_elements*sizeof(ulong2));

    for (uint64_t i = 0; i < number_of_elements; i++) {
        h_key_array[i].x = ((uint64_t)rand()) << 32 | (uint64_t)rand();
        h_key_array[i].y = ((uint64_t)rand()) << 32 | (uint64_t)rand();
    }
    printf("size : %lu\n", sizeof(ulong2));

    thrust::device_ptr<ulong2> th_key_array( h_key_array );

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float totalseconds = 0;
    int iterations = 1;
    for(int i = 0; i < iterations; i++)
    {
        hipEventRecord(start, 0);
        //thrust::sort_by_key( th_key_array, th_key_array+number_of_elements, th_value_array );
        thrust::sort( th_key_array, th_key_array+number_of_elements );
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        totalseconds = totalseconds + milliseconds;
        if(i == iterations - 1) break;
    }
    printf("Elapsed time: %f s.\n", totalseconds/(iterations*1000));


    return 0;
}

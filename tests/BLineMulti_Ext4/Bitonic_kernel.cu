#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstring>
#include "bitonic.hxx"

#define start_index_s0 2*i*batch_size
#define start_index_s1 2*i*batch_size+batch_size
#define start_index_s2 2*i*batch_size+2*batch_size

#define merge_index_1 2*(i-1)*batch_size
#define merge_index_2 2*(i-1)*batch_size+batch_size

void PairMerge(uint64_t *key_array_1, uint64_t *key_array_2, uint64_t batch_size, int nthreads);

void BitonicSort(uint64_t *h_key_array, uint64_t *d_key_array[2], uint64_t number_of_elements, uint64_t batch_size, int nthreads)
{
    int number_of_batches = number_of_elements / batch_size;
    
    uint64_t *pinned_M[2];

    hipMalloc( (void**)&d_key_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_key_array[1], batch_size * sizeof(uint64_t) );
    hipHostAlloc( (void**)&pinned_M[0], batch_size * sizeof(uint64_t), hipHostMallocDefault );
    hipHostAlloc( (void**)&pinned_M[1], batch_size * sizeof(uint64_t), hipHostMallocDefault );
    
    hipStream_t streams[2];
    for (int s = 0; s < 2; s++) {
        hipStreamCreate(&streams[s]);
    }
    
    for (int i = 0; i < number_of_batches / 2; i++) {
        for (int s = 0; s < 2; s++) {
            if (i == 0 && s == 0) {
                std::memcpy(pinned_M[0],
                            &h_key_array[start_index_s0],
                            batch_size*sizeof(uint64_t));

                hipMemcpyAsync(d_key_array[0],
                                pinned_M[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipDeviceSynchronize();
            }
            if (s == 0)
            {
                //bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
                std::memcpy(pinned_M[1],
                            &h_key_array[start_index_s1],
                            batch_size*sizeof(uint64_t));
                
                hipMemcpyAsync(d_key_array[1],
                                pinned_M[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[1]);
                bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
                if (i > 0)
                {
                    PairMerge(&h_key_array[merge_index_1], &h_key_array[merge_index_2], batch_size, nthreads);
                }
                //bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
                hipDeviceSynchronize();
            }
            else if (s == 1)
            {
                //bitonicSort<uint64_t, cmp>(d_key_array[1], batch_size, 256, 32, streams[1]);
                hipMemcpyAsync(pinned_M[0],
                                d_key_array[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[0]);
                bitonicSort<uint64_t, cmp>(d_key_array[1], batch_size, 256, 32, streams[1]);
                hipDeviceSynchronize();
                
                std::memcpy(&h_key_array[start_index_s0],
                            pinned_M[0],
                            batch_size*sizeof(uint64_t));
            }
            if (s == 1 && i != (number_of_batches / 2) - 1) {
                std::memcpy(pinned_M[0],
                            &h_key_array[start_index_s2],
                            batch_size*sizeof(uint64_t));
                
                hipMemcpyAsync(pinned_M[1],
                                d_key_array[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
                hipMemcpyAsync(d_key_array[0],
                                pinned_M[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipDeviceSynchronize();
                
                std::memcpy(&h_key_array[start_index_s1],
                            pinned_M[1],
                            batch_size*sizeof(uint64_t));
            }
            else if (s == 1 && i == (number_of_batches / 2) - 1) {
                hipMemcpyAsync(pinned_M[1],
                                d_key_array[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
                std::memcpy(&h_key_array[start_index_s1],
                            pinned_M[1],
                            batch_size*sizeof(uint64_t));
                hipDeviceSynchronize();
            }
        }
    }
    
    for (int s = 0; s < 2; s++) {
        hipStreamDestroy(streams[s]);
    }
   
    return;
}

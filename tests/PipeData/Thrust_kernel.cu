#include <stdio.h>
#include "PipeData_Thrust.h"

void PipeDataSort(uint64_t *h_key_array, uint64_t *d_key_array[2], uint64_t number_of_elements, uint64_t batch_size, uint64_t pinned_M_size, int nstreams)
{
    int number_of_batches = number_of_elements / batch_size;
    int number_of_buffers = batch_size / pinned_M_size;
    
    uint64_t *pinned_M[2];

    hipMalloc( (void**)&d_key_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_key_array[1], batch_size * sizeof(uint64_t) );
    hipHostAlloc( (void**)&pinned_M[0], (pinned_M_size / 2) * sizeof(uint64_t), hipHostMallocDefault );
    hipHostAlloc( (void**)&pinned_M[1], (pinned_M_size / 2) * sizeof(uint64_t), hipHostMallocDefault );
    
    hipStream_t streams[nstreams];
    for (int s = 0; s < nstreams; s++) {
        hipStreamCreate(&streams[s]);
    }
    
    thrust::device_ptr<uint64_t> th_key_array0( d_key_array[0] );
    thrust::device_ptr<uint64_t> th_key_array1( d_key_array[1] );
    
    int start_index = 0;
    for (int i = 0; i < number_of_batches / 2; i++) {
        for (int s = 0; s < nstreams; s++) {
            start_index = 2*i*batch_size+s*batch_size;
            //Staged HtoD
            for (int b = 0; b < number_of_buffers; b++) {
                hipMemcpyAsync(pinned_M[s], &h_key_array[start_index+b*(pinned_M_size/2)], pinned_M_size/2, hipMemcpyHostToHost, streams[s]);
                hipStreamSynchronize(streams[s]);
                hipMemcpyAsync(&d_key_array[s][b*(pinned_M_size/2)], pinned_M[s], pinned_M_size/2, hipMemcpyHostToDevice, streams[s]);
                hipStreamSynchronize(streams[s]);
            }
            
            //Sort on GPU
            thrust::sort(thrust::cuda::par.on(streams[s]), (s == 0?th_key_array0 : th_key_array1), (s == 0?th_key_array0 : th_key_array1)+batch_size, streams[s]);
            hipStreamSynchronize(streams[s]);
            
            //Staged DtoH
            for (int b = 0; b < number_of_buffers; b++) {
                hipMemcpyAsync(pinned_M[s], &d_key_array[s][b*(pinned_M_size/2)], pinned_M_size/2, hipMemcpyDeviceToHost, streams[s]);
                hipStreamSynchronize(streams[s]);
                hipMemcpyAsync(&h_key_array[start_index+b*(pinned_M_size/2)], pinned_M[s], pinned_M_size/2, hipMemcpyHostToHost, streams[s]);
                hipStreamSynchronize(streams[s]);
            }
        }
    }
    
    for (int s = 0; s < nstreams; s++) {
        hipStreamDestroy(&streams[s]);
    }
   
    return;
}

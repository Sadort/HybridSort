#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <algorithm>

void ThrustSort(uint64_t *h_key_array, uint64_t *d_key_array, uint64_t *h_value_array, uint64_t *d_value_array, uint64_t number_of_elements, uint64_t batch_size)
{
    int number_of_batches = number_of_elements / batch_size;

    hipMalloc( (void**)&d_key_array, batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_value_array, batch_size * sizeof(uint64_t) );
    hipMemcpy( d_key_array,
                h_key_array,
                batch_size * sizeof(uint64_t),
                hipMemcpyHostToDevice );
    hipMemcpy( d_value_array,
                h_value_array,
                batch_size * sizeof(uint64_t),
                hipMemcpyHostToDevice );
    thrust::device_ptr<uint64_t> th_key_array( d_key_array );
    thrust::device_ptr<uint64_t> th_value_array( d_value_array );

    for (int i = 0; i < number_of_batches; i++)
    {
        thrust::sort_by_key( th_key_array, th_key_array+batch_size, th_value_array );
        //thrust::sort( th_key_array, th_key_array+batch_size );

        hipMemcpy( &h_key_array[i*batch_size],
                    d_key_array,
                    batch_size * sizeof(uint64_t),
                    hipMemcpyDeviceToHost );
        hipMemcpy( &h_value_array[i*batch_size],
                    d_value_array,
                    batch_size * sizeof(uint64_t),
                    hipMemcpyDeviceToHost );
        hipDeviceSynchronize();

      	if(i == number_of_batches-1)
      	    break;

      	hipMemcpy( d_key_array,
                      &h_key_array[(i+1)*batch_size],
                      batch_size * sizeof(uint64_t),
                      hipMemcpyHostToDevice );
        hipMemcpy( d_value_array,
                    &h_value_array[(i+1)*batch_size],
                    batch_size * sizeof(uint64_t),
                    hipMemcpyHostToDevice );
      	hipDeviceSynchronize();

    }

    return;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <algorithm>
#include "type.h"

void ThrustSort(ulong2 *h_key_array, ulong2 *d_key_array, uint64_t number_of_elements, uint64_t batch_size)
{
    int number_of_batches = number_of_elements / batch_size;

    hipMalloc( (void**)&d_key_array, batch_size * sizeof(ulong2) );
    hipMemcpy( d_key_array,
                h_key_array,
                batch_size * sizeof(ulong2),
                hipMemcpyHostToDevice );
    thrust::device_ptr<ulong2> th_key_array( d_key_array );

    for (int i = 0; i < number_of_batches; i++)
    {        
        //thrust::sort_by_key( th_key_array, th_key_array+number_of_elements, th_value_array );
        thrust::sort( th_key_array, th_key_array+batch_size );
        
        hipMemcpy( &h_key_array[i*batch_size],
                d_key_array,
                batch_size * sizeof(ulong2),
                hipMemcpyDeviceToHost );
        hipDeviceSynchronize();
        
        if(i == number_of_batches-1)
            break;

        hipMemcpy( d_key_array,
                &h_key_array[(i+1)*batch_size],
                batch_size * sizeof(ulong2),
                hipMemcpyHostToDevice );
        hipDeviceSynchronize();
        
    }
   
    return;
}

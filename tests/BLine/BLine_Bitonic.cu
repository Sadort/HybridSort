#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream> 
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <cmath>
#include <random>
#include "bitonic_sort.h"

using namespace std;

int main(void)
{
    uint64_t number_of_elements = 1024L*1024*1024;
    uint64_t *h_key_array = (uint64_t *)malloc(number_of_elements*sizeof(uint64_t));
    uint64_t *d_key_array;

    for (uint64_t i = 0; i < number_of_elements; i++) {
        h_key_array[i] = ((uint64_t)rand()) << 32 | (uint64_t)rand();
    }
    printf("size : %lu\n", sizeof(uint64_t));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    hipEventRecord(start, 0);

    hipMalloc( (void**)&d_key_array, number_of_elements * sizeof(uint64_t) );
    hipMemcpy( d_key_array,
                h_key_array, 
                number_of_elements * sizeof(uint64_t),
                hipMemcpyHostToDevice );

    BitonicSort<uint64_t, cmp>(d_key_array, number_of_elements, 256, 32);

    hipMemcpy( h_key_array,
                d_key_array,
                number_of_elements * sizeof(uint64_t),
                hipMemcpyDeviceToHost );

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time: %f s.\n", milliseconds/1000);

    //std::sort(h_key_ref.begin(), h_key_ref.end());
    //bool result = compareAB(h_key_array, h_key_ref);
    //printf("Test: %s\n", result == true ? "SUCCESS" : "FAIL");

    return 0;
}

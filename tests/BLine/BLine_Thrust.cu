#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream> 
#include <hip/hip_runtime.h>
#include <algorithm>

using namespace std;

int main(void)
{
    uint64_t number_of_elements = 350L*1024*1024;
    uint64_t *h_key_array = (uint64_t *)malloc(number_of_elements*sizeof(uint64_t));
    uint64_t *d_key_array;

    for (uint64_t i = 0; i < number_of_elements; i++) {
        h_key_array[i] = ((uint64_t)rand()) << 32 | (uint64_t)rand();
    }
    printf("size : %lu\n", sizeof(uint64_t));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    hipEventRecord(start, 0);

    hipMalloc( (void**)&d_key_array, number_of_elements * sizeof(uint64_t) );
    hipMemcpy( d_key_array,
                h_key_array, 
                number_of_elements * sizeof(uint64_t),
                hipMemcpyHostToDevice );

    thrust::device_ptr<uint64_t> th_key_array( d_key_array );
    
    //thrust::sort_by_key( th_key_array, th_key_array+number_of_elements, th_value_array );
    thrust::sort( th_key_array, th_key_array+number_of_elements );

    hipMemcpy( h_key_array,
                d_key_array,
                number_of_elements * sizeof(uint64_t),
                hipMemcpyDeviceToHost );

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time: %f s.\n", milliseconds/1000);

    //std::sort(h_key_ref.begin(), h_key_ref.end());
    //bool result = compareAB(h_key_array, h_key_ref);
    //printf("Test: %s\n", result == true ? "SUCCESS" : "FAIL");

    return 0;
}

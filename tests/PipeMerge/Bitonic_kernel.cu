#include <stdio.h>
#include <cstring>
#include <hip/hip_runtime.h>
#include <map>
#include <cassert>
#include "bitonic.hxx"

#define start_index_s0 2*i*batch_size
#define start_index_s1 2*i*batch_size+batch_size
#define start_index_s2 2*i*batch_size-batch_size

#define merge_index_1 2*(i-1)*batch_size
#define merge_index_2 2*(i-1)*batch_size+batch_size

void PairMerge(uint64_t *key_array_1, uint64_t *key_array_2, uint64_t batch_size, int nthreads);

void BitonicSort(uint64_t *h_key_array, uint64_t *d_key_array[2], uint64_t number_of_elements, uint64_t batch_size, uint64_t pinned_M_size, int nthreads)
{
    int number_of_batches = number_of_elements / batch_size;
    int number_of_buffers = 2 * batch_size / pinned_M_size;
    
    uint64_t *pinned_M[2];

    hipMalloc( (void**)&d_key_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_key_array[1], batch_size * sizeof(uint64_t) );
    hipHostAlloc( (void**)&pinned_M[0], (pinned_M_size / 2) * sizeof(uint64_t), hipHostMallocDefault );
    hipHostAlloc( (void**)&pinned_M[1], (pinned_M_size / 2) * sizeof(uint64_t), hipHostMallocDefault );
    
    hipStream_t streams[2];
    for (int s = 0; s < 2; s++) {
        hipStreamCreate(&streams[s]);
    }
        
    for (int i = 0; i < number_of_batches / 2; i++) {
        for (int s = 0; s < 2; s++) {
            if (i == 0 && s == 0) {
                for (int b = 0; b < number_of_buffers; b++) {
                    std::memcpy(pinned_M[0],
                                &h_key_array[start_index_s0+b*(pinned_M_size/2)],
                                (pinned_M_size/2)*sizeof(uint64_t));
                    hipStreamSynchronize(streams[0]);
                    
                    hipMemcpyAsync(&d_key_array[0][b*(pinned_M_size/2)],
                                    pinned_M[0],
                                    (pinned_M_size/2)*sizeof(uint64_t),
                                    hipMemcpyHostToDevice,
                                    streams[0]);
                    hipStreamSynchronize(streams[0]);
                }
                //thrust::sort(thrust::cuda::par(alloc).on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
                hipStreamSynchronize(streams[0]);
            }
            else if (i > 0 && s == 0) {
                //Overlapping
                for (int b = 0; b < number_of_buffers; b++) {
                    hipMemcpyAsync(pinned_M[1],
                                    &d_key_array[1][b*(pinned_M_size/2)],
                                    (pinned_M_size/2)*sizeof(uint64_t),
                                    hipMemcpyDeviceToHost,
                                    streams[1]);
                    
                    std::memcpy(pinned_M[0],
                                &h_key_array[start_index_s0+b*(pinned_M_size/2)],
                                (pinned_M_size/2)*sizeof(uint64_t));
                    hipStreamSynchronize(streams[1]);
                    
                    hipMemcpyAsync(&d_key_array[0][b*(pinned_M_size/2)],
                                    pinned_M[0],
                                    (pinned_M_size/2)*sizeof(uint64_t),
                                    hipMemcpyHostToDevice,
                                    streams[0]);
                    std::memcpy(&h_key_array[start_index_s2+b*(pinned_M_size/2)],
                                pinned_M[1],
                                (pinned_M_size/2)*sizeof(uint64_t));
                    hipStreamSynchronize(streams[0]);
                }
                //thrust::sort(thrust::cuda::par(alloc).on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
                PairMerge(&h_key_array[merge_index_1], &h_key_array[merge_index_2], batch_size, nthreads);
                hipStreamSynchronize(streams[0]);
            }
            else if (s == 1) {
                //Overlapping
                for (int b = 0; b < number_of_buffers; b++) {
                    hipMemcpyAsync(pinned_M[0],
                                    &d_key_array[0][b*(pinned_M_size/2)],
                                    (pinned_M_size/2)*sizeof(uint64_t),
                                    hipMemcpyDeviceToHost,
                                    streams[0]);
                    std::memcpy(pinned_M[1],
                                &h_key_array[start_index_s1+b*(pinned_M_size/2)],
                                (pinned_M_size/2)*sizeof(uint64_t));
                    hipStreamSynchronize(streams[0]);
                    
                    hipMemcpyAsync(&d_key_array[1][b*(pinned_M_size/2)],
                                    pinned_M[1],
                                    (pinned_M_size/2)*sizeof(uint64_t),
                                    hipMemcpyHostToDevice,
                                    streams[1]);
                    std::memcpy(&h_key_array[start_index_s0+b*(pinned_M_size/2)],
                                pinned_M[0],
                                (pinned_M_size/2)*sizeof(uint64_t));
                    hipStreamSynchronize(streams[1]);
                }
                //thrust::sort(thrust::cuda::par(alloc).on(streams[1]), th_key_array[1], th_key_array[1]+batch_size);
                bitonicSort<uint64_t, cmp>(d_key_array[1], batch_size, 256, 32, streams[1]);
                hipStreamSynchronize(streams[1]);
                    
                if (i == (number_of_batches / 2) - 1) {
                    for (int b = 0; b < number_of_buffers; b++) {
                        hipMemcpyAsync(pinned_M[1],
                                        &d_key_array[1][b*(pinned_M_size/2)],
                                        (pinned_M_size/2)*sizeof(uint64_t),
                                        hipMemcpyDeviceToHost,
                                        streams[1]);
                        hipStreamSynchronize(streams[1]);
                        
                        std::memcpy(&h_key_array[start_index_s1+b*(pinned_M_size/2)],
                                    pinned_M[1],
                                    (pinned_M_size/2)*sizeof(uint64_t));
                        hipStreamSynchronize(streams[1]);
                    }
                }
            }
            
            
        }
        
    }
    
    for (int s = 0; s < 2; s++) {
        hipStreamDestroy(streams[s]);
    }
   
    return;
}

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "bitonic.hxx"

#define start_index_s0 2*i*batch_size
#define start_index_s1 2*i*batch_size+batch_size
#define start_index_s2 2*i*batch_size-batch_size

void BitonicSort(uint64_t *h_key_array, uint64_t *d_key_array[2], uint64_t number_of_elements, uint64_t batch_size, int nstreams = 2)
{
    int number_of_batches = number_of_elements / batch_size;

    hipMalloc( (void**)&d_key_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_key_array[1], batch_size * sizeof(uint64_t) );
    
    hipStream_t streams[2];
    for (int s = 0; s < 2; s++) {
        hipStreamCreate(&streams[s]);
    }
    
    for (int i = 0; i < number_of_batches / 2; i++) {
        for (int s = 0; s < 2; s++) {
            if (i == 0 && s == 0) {
		hipMemcpyAsync(d_key_array[0],
                               &h_key_array[start_index_s0],
                               (batch_size)*sizeof(uint64_t),
                               hipMemcpyHostToDevice,
               		       streams[0]);
                //thrust::sort(thrust::cuda::par.on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
		hipStreamSynchronize(streams[0]);
            }
            else if (i > 0 && s == 0) {
                //Overlapping
		hipMemcpyAsync(&h_key_array[start_index_s2],
                                d_key_array[1],
                                (batch_size)*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
		hipMemcpyAsync(d_key_array[0],
                                &h_key_array[start_index_s0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
		hipDeviceSynchronize();
                //thrust::sort(thrust::cuda::par.on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
		hipStreamSynchronize(streams[0]);
            }
            else if (s == 1) {
                //Overlapping
                hipMemcpyAsync(&h_key_array[start_index_s0],
                                d_key_array[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[0]);
                hipMemcpyAsync(d_key_array[1],
                                &h_key_array[start_index_s1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[1]);
		hipDeviceSynchronize();
                //thrust::sort(thrust::cuda::par.on(streams[1]), th_key_array[1], th_key_array[1]+batch_size);
                bitonicSort<uint64_t, cmp>(d_key_array[1], batch_size, 256, 32, streams[1]);
		hipStreamSynchronize(streams[1]);
                
                if (i == (number_of_batches / 2) - 1) {
		    hipMemcpyAsync(&h_key_array[start_index_s1],
                                    d_key_array[1],
                                    batch_size*sizeof(uint64_t),
                                    hipMemcpyDeviceToHost,
                                    streams[1]);
		    hipStreamSynchronize(streams[1]);
                }
            }
            
            
        }
        
    }
    
    for (int s = 0; s < 2; s++) {
        hipStreamDestroy(streams[s]);
    }
   
    return;
}

#include <stdio.h>
#include <stdlib.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <algorithm>

#define start_index_s0 2*i*batch_size
#define start_index_s1 2*i*batch_size+batch_size
#define start_index_s2 2*i*batch_size-batch_size

void ThrustSort(uint64_t *h_key_array, uint64_t *d_key_array[2], uint64_t number_of_elements, uint64_t batch_size, int nstreams = 2)
{
    int number_of_batches = number_of_elements / batch_size;
    
    uint64_t *pinned_M[2];

    hipMalloc( (void**)&d_key_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_key_array[1], batch_size * sizeof(uint64_t) );
    hipHostAlloc( (void**)&pinned_M[0], batch_size * sizeof(uint64_t), hipHostMallocDefault );
    hipHostAlloc( (void**)&pinned_M[1], batch_size * sizeof(uint64_t), hipHostMallocDefault );
    
    hipStream_t streams[2];
    for (int s = 0; s < 2; s++) {
        hipStreamCreate(&streams[s]);
    }
    
    thrust::device_ptr<uint64_t> th_key_array[2];
    for (int s = 0; s < 2; s++) {
        th_key_array[s] = thrust::device_pointer_cast(d_key_array[s]);
    }
    
    for (int i = 0; i < number_of_batches / 2; i++) {
        for (int s = 0; s < 2; s++) {
            if (i == 0 && s == 0) {
                hipMemcpyAsync(d_key_array[0],
                                &h_key_array[start_index_s0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipDeviceSynchronize();
                thrust::sort(thrust::cuda::par.on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                hipDeviceSynchronize();
            }
            else if (i > 0 && s == 0) {
                //Overlapping
                std::memcpy(pinned_M[0],
                            &h_key_array[start_index_s0],
                            batch_size*sizeof(uint64_t));
                
                hipMemcpyAsync(pinned_M[1],
                                d_key_array[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
                hipMemcpyAsync(d_key_array[0],
                                pinned_M[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipDeviceSynchronize();
                
                std::memcpy(&h_key_array[start_index_s2],
                            pinned_M[1],
                            batch_size*sizeof(uint64_t));
                
                thrust::sort(thrust::cuda::par.on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                hipDeviceSynchronize();
            }
            else if (s == 1) {
                //Overlapping
                std::memcpy(pinned_M[1],
                            &h_key_array[start_index_s1],
                            batch_size*sizeof(uint64_t));
                
                hipMemcpyAsync(&pinned_M[0],
                                d_key_array[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[0]);
                hipMemcpyAsync(d_key_array[1],
                                &pinned_M[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[1]);
                hipDeviceSynchronize();
                
                std::memcpy(&h_key_array[start_index_s0],
                            pinned_M[0],
                            batch_size*sizeof(uint64_t));
                
                thrust::sort(thrust::cuda::par.on(streams[1]), th_key_array[1], th_key_array[1]+batch_size);
                hipDeviceSynchronize();
                
                if (i == (number_of_batches / 2) - 1) {
                    hipMemcpyAsync(&h_key_array[start_index_s1],
                                    d_key_array[1],
                                    batch_size*sizeof(uint64_t),
                                    hipMemcpyDeviceToHost,
                                    streams[1]);
                    hipDeviceSynchronize();
                }
            }
            
            
        }
        
    }
    
    for (int s = 0; s < 2; s++) {
        hipStreamDestroy(streams[s]);
    }
   
    return;
}

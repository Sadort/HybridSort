#include <stdio.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/vector.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/pair.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <map>
#include <cassert>

#define start_index_s0 2*i*batch_size
#define start_index_s1 2*i*batch_size+batch_size
#define start_index_s2 2*i*batch_size+2*batch_size

// cached_allocator: a simple allocator for caching allocation requests
class cached_allocator
{
  public:
    // just allocate bytes
    typedef char value_type;

    cached_allocator() {}

    ~cached_allocator()
    {
      // free all allocations when cached_allocator goes out of scope
      free_all();
    }

    char *allocate(std::ptrdiff_t num_bytes)
    {
      char *result = 0;

      // search the cache for a free block
      free_blocks_type::iterator free_block = free_blocks.find(num_bytes);

      if(free_block != free_blocks.end())
      {
        std::cout << "cached_allocator::allocator(): found a hit" << std::endl;

        // get the pointer
        result = free_block->second;

        // erase from the free_blocks map
        free_blocks.erase(free_block);
      }
      else
      {
        // no allocation of the right size exists
        // create a new one with cuda::malloc
        // throw if cuda::malloc can't satisfy the request
        try
        {
          std::cout << "cached_allocator::allocator(): no free block found; calling cuda::malloc" << std::endl;

          // allocate memory and convert cuda::pointer to raw pointer
          result = thrust::cuda::malloc<char>(num_bytes).get();
        }
        catch(std::runtime_error &e)
        {
          throw;
        }
      }

      // insert the allocated pointer into the allocated_blocks map
      allocated_blocks.insert(std::make_pair(result, num_bytes));

      return result;
    }

    void deallocate(char *ptr, size_t n)
    {
      // erase the allocated block from the allocated blocks map
      allocated_blocks_type::iterator iter = allocated_blocks.find(ptr);
      std::ptrdiff_t num_bytes = iter->second;
      allocated_blocks.erase(iter);

      // insert the block into the free blocks map
      free_blocks.insert(std::make_pair(num_bytes, ptr));
    }

  private:
    typedef std::multimap<std::ptrdiff_t, char*> free_blocks_type;
    typedef std::map<char *, std::ptrdiff_t>     allocated_blocks_type;

    free_blocks_type      free_blocks;
    allocated_blocks_type allocated_blocks;

    void free_all()
    {
      std::cout << "cached_allocator::free_all(): cleaning up after ourselves..." << std::endl;

      // deallocate all outstanding blocks in both lists
      for(free_blocks_type::iterator i = free_blocks.begin();
          i != free_blocks.end();
          ++i)
      {
        // transform the pointer to cuda::pointer before calling cuda::free
        thrust::cuda::free(thrust::cuda::pointer<char>(i->second));
      }

      for(allocated_blocks_type::iterator i = allocated_blocks.begin();
          i != allocated_blocks.end();
          ++i)
      {
        // transform the pointer to cuda::pointer before calling cuda::free
        thrust::cuda::free(thrust::cuda::pointer<char>(i->first));
      }
    }

};

void ThrustSort(uint64_t *h_key_array, uint64_t *d_key_array[2], uint64_t *h_value_array, uint64_t *d_value_array[2], uint64_t number_of_elements, uint64_t batch_size)
{
    cached_allocator alloc;
    int number_of_batches = number_of_elements / batch_size;

    uint64_t *pinned_key_M[2];
    uint64_t *pinned_value_M[2];

    hipMalloc( (void**)&d_key_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_key_array[1], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_value_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_value_array[1], batch_size * sizeof(uint64_t) );
    hipHostAlloc( (void**)&pinned_key_M[0], batch_size * sizeof(uint64_t), hipHostMallocDefault );
    hipHostAlloc( (void**)&pinned_key_M[1], batch_size * sizeof(uint64_t), hipHostMallocDefault );
    hipHostAlloc( (void**)&pinned_value_M[0], batch_size * sizeof(uint64_t), hipHostMallocDefault );
    hipHostAlloc( (void**)&pinned_value_M[1], batch_size * sizeof(uint64_t), hipHostMallocDefault );

    hipStream_t streams[2];
    for (int s = 0; s < 2; s++) {
        hipStreamCreate(&streams[s]);
    }

    thrust::device_ptr<uint64_t> th_key_array[2];
    thrust::device_ptr<uint64_t> th_value_array[2];
    for (int s = 0; s < 2; s++) {
        th_key_array[s] = thrust::device_pointer_cast(d_key_array[s]);
        th_value_array[s] = thrust::device_pointer_cast(d_value_array[s]);
    }

    for (int i = 0; i < number_of_batches / 2; i++) {
        for (int s = 0; s < 2; s++) {
            if (i == 0 && s == 0) {
                hipMemcpyAsync(d_key_array[0],
                                &h_key_array[start_index_s0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipMemcpyAsync(d_value_array[0],
                                &h_value_array[start_index_s0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipDeviceSynchronize();
            }
            if (s == 0)
            {
                //thrust::sort(thrust::cuda::par(alloc).on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                std::memcpy(pinned_key_M[1],
                            &h_key_array[start_index_s1],
                            batch_size*sizeof(uint64_t));
                std::memcpy(pinned_value_M[1],
                            &h_value_array[start_index_s1],
                            batch_size*sizeof(uint64_t));
                hipMemcpyAsync(d_key_array[1],
                                pinned_key_M[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[1]);
                hipMemcpyAsync(d_value_array[1],
                                pinned_value_M[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[1]);
                thrust::sort_by_key(thrust::cuda::par(alloc).on(streams[0]), th_key_array[0], th_key_array[0]+batch_size, th_value_array[0]);
                hipDeviceSynchronize();
            }
            else if (s == 1)
            {
                //thrust::sort(thrust::cuda::par(alloc).on(streams[1]), th_key_array[1], th_key_array[1]+batch_size);
                hipMemcpyAsync(pinned_key_M[0],
                                d_key_array[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[0]);
                hipMemcpyAsync(pinned_value_M[0],
                                d_value_array[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[0]);
                thrust::sort_by_key(thrust::cuda::par(alloc).on(streams[1]), th_key_array[1], th_key_array[1]+batch_size, th_value_array[1]);
                hipDeviceSynchronize();

                std::memcpy(&h_key_array[start_index_s0],
                            pinned_key_M[0],
                            batch_size*sizeof(uint64_t));
                std::memcpy(&h_value_array[start_index_s0],
                            pinned_value_M[0],
                            batch_size*sizeof(uint64_t));
            }
            if (s == 1 && i != (number_of_batches / 2) - 1) {
                std::memcpy(pinned_key_M[0],
                            &h_key_array[start_index_s2],
                            batch_size*sizeof(uint64_t));
                std::memcpy(pinned_value_M[0],
                            &h_value_array[start_index_s2],
                            batch_size*sizeof(uint64_t));

                hipMemcpyAsync(pinned_key_M[1],
                                d_key_array[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
                hipMemcpyAsync(d_key_array[0],
                                pinned_key_M[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipMemcpyAsync(pinned_value_M[1],
                                d_value_array[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
                hipMemcpyAsync(d_value_array[0],
                                pinned_value_M[0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
                hipDeviceSynchronize();

                std::memcpy(&h_key_array[start_index_s1],
                            pinned_key_M[1],
                            batch_size*sizeof(uint64_t));
                std::memcpy(&h_value_array[start_index_s1],
                            pinned_value_M[1],
                            batch_size*sizeof(uint64_t));
            }
            else if (s == 1 && i == (number_of_batches / 2) - 1) {
                hipMemcpyAsync(&h_key_array[start_index_s1],
                                d_key_array[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
                hipMemcpyAsync(&h_value_array[start_index_s1],
                                d_value_array[1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
                hipDeviceSynchronize();
            }
        }
    }

    for (int s = 0; s < 2; s++) {
        hipStreamDestroy(streams[s]);
    }

    return;
}

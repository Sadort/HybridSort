#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "bitonic.hxx"

#define start_index_s0 2*i*batch_size
#define start_index_s1 2*i*batch_size+batch_size
#define start_index_s2 2*i*batch_size-batch_size

void BitonicSort(uint64_t *h_key_array, uint64_t *d_key_array[2], uint64_t number_of_elements, uint64_t batch_size, int nstreams = 2)
{
    int number_of_batches = number_of_elements / batch_size;

    hipMalloc( (void**)&d_key_array[0], batch_size * sizeof(uint64_t) );
    hipMalloc( (void**)&d_key_array[1], batch_size * sizeof(uint64_t) );
    
    hipStream_t streams[2];
    for (int s = 0; s < 2; s++) {
        hipStreamCreate(&streams[s]);
    }
    
    for (int i = 0; i < number_of_batches / 2; i++) {
        for (int s = 0; s < 2; s++) {
            if (i == 0 && s == 0) {
		hipMemcpyAsync(d_key_array[0],
                               &h_key_array[start_index_s0],
                               batch_size*sizeof(uint64_t),
                               hipMemcpyHostToDevice,
               		       streams[0]);
		hipDeviceSynchronize();
            }
	    if (s == 0)
	    {
	        //thrust::sort(thrust::cuda::par(alloc).on(streams[0]), th_key_array[0], th_key_array[0]+batch_size);
                bitonicSort<uint64_t, cmp>(d_key_array[0], batch_size, 256, 32, streams[0]);
		hipMemcpyAsync(d_key_array[1],
                                &h_key_array[start_index_s1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[1]);
		hipDeviceSynchronize();
	    }
	    else if (s == 1)
	    {
	        //thrust::sort(thrust::cuda::par(alloc).on(streams[1]), th_key_array[1], th_key_array[1]+batch_size);
		bitonicSort<uint64_t, cmp>(d_key_array[1], batch_size, 256, 32, streams[1]);
		hipMemcpyAsync(d_key_array[0],
                                &h_key_array[start_index_s0],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[0]);
	        hipDeviceSynchronize();
	    }
            if (s == 1 && i != (number_of_batches / 2) - 1) {
	        hipMemcpyAsync(d_key_array[1],
                                &h_key_array[start_index_s1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
		hipMemcpyAsync(d_key_array[0],
                                &h_key_array[start_index_s2],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyHostToDevice,
                                streams[0]);
		hipDeviceSynchronize();
	    }
	    else if (s == 1 && i == (number_of_batches / 2) - 1) {
	        hipMemcpyAsync(d_key_array[1],
                                &h_key_array[start_index_s1],
                                batch_size*sizeof(uint64_t),
                                hipMemcpyDeviceToHost,
                                streams[1]);
	        hipDeviceSynchronize();
	    }
        }
    }
    
    for (int s = 0; s < 2; s++) {
        hipStreamDestroy(streams[s]);
    }
   
    return;
}
